#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */


#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvToolsExt.h>
#include <thrust/host_vector.h>
#include <tuple>
#include "types.h"
#include "runtime_maintenance.h"

__device__ int safeexc(int* data, int index)
{
	if (index == 0)
		return 0;
	return data[index - 1];
}

__global__ void rearrange(
	int N,
	int* active_nodes,
	int* new_active_nodes,
	const Node* nodes,
	const Box* boxes,
	const float3* pos_space,
	const float4* rot_space,
	const float* sh_space,
	const float* alpha_space,
	const float3* scale_space,
	int* split_space,
	Node* new_nodes,
	Box* new_boxes,
	float3* new_pos_space,
	float4* new_rot_space,
	float* new_sh_space,
	float* new_alpha_space,
	float3* new_scale_space,
	int* new_split_space,
	int* cuda2cpu_src,
	int* cuda2cpu_dst,
	int* node_indices,
	int* gaussian_indices)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if ((idx/15) >= N)
		return;

	int role = idx / N;
	idx = idx % N;
	bool actor = role == 0;

	int node_id = active_nodes[idx];
	int target_id = safeexc(node_indices, node_id);

	Node node = nodes[node_id];

	if (actor)
	{
		new_boxes[target_id] = boxes[node_id];
		new_split_space[target_id] = split_space[node_id];

		if (split_space[node_id] == 0) // Every unexpanded node is gone
			node.start_children = -1;
		split_space[node_id] = 0; // Clean up after yourself

		int new_parent = node.parent == -1 ? -1 : safeexc(node_indices, node.parent);
		int new_start_children = node.start_children == -1 ? -1 : safeexc(node_indices, node.start_children);

		node.parent = new_parent;
		node.start_children = new_start_children;
	}

	int new_start = safeexc(gaussian_indices, node_id);
	for (int i = 0; i < node.count_leafs + node.count_merged; i++)
	{
		int dst = new_start + i;
		int src = node.start + i;
		if (role == 0)
			new_pos_space[dst] = pos_space[src];
		else if (role == 1)
			new_rot_space[dst] = rot_space[src];
		else if (role >= 2 && role < 14)
			*(((float4*)(new_sh_space + dst * 48)) + role - 2) = *(((float4*)(sh_space + src * 48)) + role - 2);
		else
		{
			new_alpha_space[dst] = alpha_space[src];
			new_scale_space[dst] = scale_space[src];
		}
	}

	if (actor)
	{
		node.start = new_start;
		new_nodes[target_id] = node;
		new_active_nodes[idx] = idx;
		cuda2cpu_dst[target_id] = cuda2cpu_src[node_id];
	}
}

void Maintenance::reorder(
	int N,
	int* active_nodes,
	int* new_active_nodes,
	const int* nodes,
	const float* boxes,
	const float* pos_space,
	const float* rot_space,
	const float* sh_space,
	const float* alpha_space,
	const float* scale_space,
	int* split_space,
	int* new_nodes,
	float* new_boxes,
	float* new_pos_space,
	float* new_rot_space,
	float* new_sh_space,
	float* new_alpha_space,
	float* new_scale_space,
	int* new_split_space,
	int* cuda2cpu_src,
	int* cuda2cpu_dst,
	int* node_indices,
	int* gaussian_indices,
	void* streamy
)
{
	hipStream_t stream = (hipStream_t)streamy;
	int num_blocks = (N * 15 + 255) / 256;
	rearrange << <num_blocks, 256, 0, stream >> > (
		N,
		active_nodes,
		new_active_nodes,
		(Node*)nodes,
		(Box*)boxes,
		(float3*)pos_space,
		(float4*)rot_space,
		sh_space,
		alpha_space,
		(float3*)scale_space,
		split_space,
		(Node*)new_nodes,
		(Box*)new_boxes,
		(float3*)new_pos_space,
		(float4*)new_rot_space,
		new_sh_space,
		new_alpha_space,
		(float3*)new_scale_space,
		new_split_space,
		cuda2cpu_src,
		cuda2cpu_dst,
		node_indices,
		gaussian_indices
		);
}

__global__ void mark(
	int N,
	const int* indices,
	Node* nodes,
	int* nodes_count,
	int* gaussians_count)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	nodes_count[node_id] = 1;
	gaussians_count[node_id] = node.count_merged + node.count_leafs;
}

__global__ void zero(int N, int* a, int* b)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= N)
		return;

	a[idx] = 0;
	b[idx] = 0;
}

void Maintenance::compactPart1(
	int topN,
	int N,
	int* active_nodes,
	int* new_active_nodes,
	const int* nodes,
	const float* boxes,
	const float* pos_space,
	const float* rot_space,
	const float* sh_space,
	const float* alpha_space,
	const float* scale_space,
	int* split_space,
	int* new_nodes,
	float* new_boxes,
	float* new_pos_space,
	float* new_rot_space,
	float* new_sh_space,
	float* new_alpha_space,
	float* new_scale_space,
	int* new_split_space,
	int* cuda2cpu_src,
	int* cuda2cpu_dst,
	int* NsrcI,
	int* NsrcI2,
	int* NdstI,
	int* NdstI2,
	char*& scratchspace,
	size_t& scratchspacesize,
	void* streamy,
	int* count
)
{
	hipStream_t stream = (hipStream_t)streamy;

	zero << <(topN + 255) / 256, 256, 0, stream >> > (topN, NsrcI, NsrcI2);
	mark << <(N + 255) / 256, 256, 0, stream >> > (N, active_nodes, (Node*)nodes, NsrcI, NsrcI2);

	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, NsrcI, NdstI, topN, stream);
	hipMemcpyAsync(count, NdstI + topN - 1, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
}

__global__ void compressCUDA(
	int N,
	Node* nodes,
	float3* scales,
	float4* rots,
	float* shs,
	float* opacities
)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= N)
		return;

	Node node = nodes[idx];

	int g = node.start;

	int parent = g;
	if (node.parent != -1)
	{
		parent = nodes[node.parent].start;
	}

	__half2 mp;
	float* scale = (float*)&scales[g];
	float* pscale = (float*)&scales[parent];
	for (int i = 0; i < 3; i++)
	{
		mp.x = scale[i];
		mp.y = pscale[i];
		scale[i] = *((float*)&mp); // We are overwriting ourselves
	}
}

void Maintenance::compress(
	int N,
	int* nodes,
	float* scales,
	float* rots,
	float* shs,
	float* opacs
)
{
	int num_blocks = (N + 255) / 256;
	compressCUDA << <num_blocks, 256 >> > (N,
		(Node*)nodes,
		(float3*)scales,
		(float4*)rots,
		shs,
		opacs
		);
}

void Maintenance::compactPart2(
	int topN,
	int N,
	int* active_nodes,
	int* new_active_nodes,
	const int* nodes,
	const float* boxes,
	const float* pos_space,
	const float* rot_space,
	const float* sh_space,
	const float* alpha_space,
	const float* scale_space,
	int* split_space,
	int* new_nodes,
	float* new_boxes,
	float* new_pos_space,
	float* new_rot_space,
	float* new_sh_space,
	float* new_alpha_space,
	float* new_scale_space,
	int* new_split_space,
	int* cuda2cpu_src,
	int* cuda2cpu_dst,
	int* NsrcI,
	int* NsrcI2,
	int* NdstI,
	int* NdstI2,
	char*& scratchspace,
	size_t& scratchspacesize,
	void* streamy,
	int* count
)
{
	hipStream_t stream = (hipStream_t)streamy;

	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, NsrcI2, NdstI2, topN, stream);

	hipMemcpyAsync(count, NdstI2 + topN - 1, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);

	reorder(
		N,
		active_nodes,
		new_active_nodes,
		nodes,
		boxes,
		pos_space,
		rot_space,
		sh_space,
		alpha_space,
		scale_space,
		split_space,
		new_nodes,
		new_boxes,
		new_pos_space,
		new_rot_space,
		new_sh_space,
		new_alpha_space,
		new_scale_space,
		new_split_space,
		cuda2cpu_src,
		cuda2cpu_dst,
		NdstI,
		NdstI2,
		stream
	);
}

__global__ void setStarts(Node* nodes, int N, int* indices, int* starts)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= N)
		return;

	int parent_id = indices[idx];
	nodes[parent_id].start_children = starts[idx];
}

void Maintenance::updateStarts(
	int* nodes,
	int num_indices,
	int* indices,
	int* starts,
	void* streamy
)
{
	hipStream_t stream = (hipStream_t)streamy;
	int num_blocks = (num_indices + 255) / 256;
	setStarts << <num_blocks, 256, 0, stream >> > ((Node*)nodes, num_indices, indices, starts);
}